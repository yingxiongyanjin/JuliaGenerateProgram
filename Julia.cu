#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

// 复数
struct hipComplex
{
    /* data */
    float r;
    float i;

    // 初始化方法
    __device__ hipComplex(float a, float b):r(a),i(b) {}

    // 计算复数的模
    __device__ float magnitude2(void) {return r*r + i*i;}

    // 规定复数的乘法和加法
    __device__ hipComplex operator*(const hipComplex &a){
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex &a){
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__
int julia
(
    int x, int y,
    int DIM,
    float scale, int step, float inf
){
    // 首先通过缩放、平移将x和y的值控制在[-scale,scale]之间
    // 生成复数平面上的一个点
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8,0.156);               // 迭代式中的常量，通过改变它可以获得不同的Julia集图案
    hipComplex a(jx, jy);

    // 进行迭代，判断点集是否发散
    for(int i=0; i<step; i++){
        a = a*a + c;
        if(a.magnitude2() > inf)
            return 0;
    }

    return 1;
}

__global__
void kernel( 
    int *bitmap,
    int DIM,
    float scale, int step, float inf
)
{
    for(int y = 0; y < DIM; y++){
        for(int x = 0; x < DIM; x++){

            // 判断是否为Julia集中的点
            int juliaValue = julia(x, y, DIM, scale,step,inf);

            // 如果是Julia集中的点，将其标记为白色
            // 如果不是Julia集中的点，则将其标记为红色
            int offset = x + y * DIM;
            bitmap[offset] = 255 * juliaValue;
        }
    }
}

void checkOut(
    int &sum,
    int *bitmap,int DIM
)
{
    sum = 0;
    for (int i = 0; i < DIM; i++) {
        for (int j = 0; j < DIM; j++) {
            if(bitmap[i*DIM+j] != 0 ){
                sum++;
            }
        }
    }
}

int main( void ){

    clock_t start, end;
    double cpu_time_used;
    start = clock(); // 记录开始时间

    // 本程序的控制参数列表
    const int DIM = 10000;                 // 二维数组的大小
    int *bitmap;                           // 存储位图数据,采用灰度图存储结果

    const float scale = 1.5;               // 控制x,y的值域，具体的说先通过 f(x) = 1 - 2x/DIM 将值映射到[-1,1],再通过该因子进行放缩
    const int step = 200;                  // 控制迭代次数
    const float inf = 1000;                // 判断点是否发散的设置的无穷大值
    
    // 动态分配空间
    int n = DIM * DIM;
    const int size = n*sizeof(int);

    // CUDA内存管理办法
    hipMallocManaged(&bitmap, size);

    // 获取设备信息
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    // 核心代码
    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    kernel<<<numberOfBlocks,threadsPerBlock>>>(bitmap,DIM,scale,step,inf);

    hipError_t Err;
    hipError_t asyncErr;

    Err = hipGetLastError();
    if(Err != hipSuccess) printf("Error: %s\n", hipGetErrorString(Err));

    // 阻断，并进行同步
    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    // 预取回主机
    hipMemPrefetchAsync(bitmap, size, hipCpuDeviceId);

    int sum = 0;
    checkOut(sum,bitmap,DIM);

    // 释放内存
    hipFree(bitmap);

    printf("done! %d\n",sum);

    end = clock(); // 记录结束时间
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("程序运行时间为 %f 秒\n", cpu_time_used);

    // 程序结束
    return 0;
}